#include "hip/hip_runtime.h"
//
// Created by erick on 5/8/25.
//
#include "nbody.h"
#include <hip/hip_runtime.h>
#define G_CONSTANT 6.67430e-11f
#define INT_STEP 0.01f
#define NEAR_ZERO 1e-10f

// universal gravitational constant
const float G = G_CONSTANT;
// integration step
const float dt = INT_STEP;


/**
 * CUDA kernel
 * bodies: pointer to bodies array
 * n number of bodies
 * **/
__global__ void updateBodies(Body* bodies, int n) {
    // i is the body index (global thread index),
    // each thread handles ONE BODY
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // index can go no longer than the number of bodies
    if (i >= n) return;

    // for this body
    Body bi = bodies[i];

    // border conditions, initial net force is null
    float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;

    // for each other body
    for (int j = 0; j < n; ++j) {
        // skip self
        if (i == j) continue;
        // this other body
        Body bj = bodies[j];

        // the distance between bodies in x, y and z
        float dx = bj.x - bi.x;
        float dy = bj.y - bi.y;
        float dz = bj.z - bi.z;

        // euclidean distance (avoid division by zero by adding a small constant)
        float distSqr = dx * dx + dy * dy + dz * dz + NEAR_ZERO;
        // inverse of the distance
        float invDist = rsqrtf(distSqr);

        // Newton's gravity, vectorial form
        float F = G * bi.mass * bj.mass * powf(invDist, 3.0f);

        // update net force over body for x,y,z
        Fx += F * dx;
        Fy += F * dy;
        Fz += F * dz;
    }

    /** update velocity
     * if (F = m * a) and (a =  dv/dt)
     * then (F = m * dv/dt)
     * then (dv = F * dt / m)
     * then v = v + dv
     * **/
    bi.vx += Fx / bi.mass * dt;
    bi.vy += Fy / bi.mass * dt;
    bi.vz += Fz / bi.mass * dt;

    /** update position
     * v = dx/dt
     * dx = dv * dt
     * x = x + dx
     **/
    bi.x += bi.vx * dt;
    bi.y += bi.vy * dt;
    bi.z += bi.vz * dt;

    // store the body back into GLOBAL MEMORY
    bodies[i] = bi;
}

/**
 * Host Function (CPU-side)
 * Copy the input data from host memory to device memory, also known as host-to-device transfer.
 * Load the GPU program and execute, caching data on-chip for performance.
 * Copy the results from device memory to host memory, also called device-to-host transfer.
 * source: https://developer.nvidia.com/blog/cuda-refresher-cuda-programming-model/

 * @param h_bodies
 * @param n number of bodies
 * @param steps simulation steps
 */
void simulateNBodyCUDA(Body* h_bodies, int n, int steps) {
    // destination memory address pointer
    Body* d_bodies;
    // in memory size of n bodies
    size_t size = n * sizeof(Body);

    // allocate GPU memory
    hipMalloc(&d_bodies, size);

    // copy data between host and device
    hipMemcpy(d_bodies, h_bodies, size, hipMemcpyHostToDevice);

    // configure threads per block
    int threadsPerBlock = 256;
    // The total number of blocks is the data size divided by the size of each block
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // for each step
    for (int s = 0; s < steps; ++s) {
        // kernel launch
        updateBodies<<<numBlocks, threadsPerBlock>>>(d_bodies, n);
        // necesary to exchange info between streams
        hipDeviceSynchronize();
    }

    // retrieve the updated positions and velocities
    hipMemcpy(h_bodies, d_bodies, size, hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_bodies);
}