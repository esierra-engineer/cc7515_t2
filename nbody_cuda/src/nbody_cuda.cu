//
// Created by erick on 5/8/25.
//
#include "include/nbody.h"
#include <hip/hip_runtime.h>
#include "include/utils.h"

/**
 * Host Function (CPU-side)
 * Copy the input data from host memory to device memory, also known as host-to-device transfer.
 * Load the GPU program and execute, caching data on-chip for performance.
 * Copy the results from device memory to host memory, also called device-to-host transfer.
 * source: https://developer.nvidia.com/blog/cuda-refresher-cuda-programming-model/

 * @param h_bodies
 * @param n number of bodies
 * @param steps simulation steps
 */
void simulateNBodyCUDA(Body* h_bodies,  int steps, float dt, const char* kernelFilename, size_t localSize, int n) {
    // destination memory address pointer
    Body* d_bodies;
    // in memory size of n bodies
    size_t size = n * sizeof(Body);

    // allocate GPU memory
    hipMalloc(&d_bodies, size);

    // copy data between host and device
    hipMemcpy(d_bodies, h_bodies, size, hipMemcpyHostToDevice);

    // configure threads per block
    size_t threadsPerBlock = localSize;
    // The total number of blocks is the data size divided by the size of each block
    size_t numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Kernel
    hipFunction_t kernel = loadKernelSource(kernelFilename);

    // for each step
    for (int s = 0; s < steps; ++s) {
        // kernel launch
        //updateBodies<<<numBlocks, threadsPerBlock>>>(d_bodies,

        // Kernel args deben ser punteros a los datos
        void* kernelArgs[] = {
            (void*) &d_bodies,
            (void*) &n,
            (void*) &dt
        };

        checkCudaErrors(
            hipModuleLaunchKernel(kernel,
            numBlocks, 1, 1,                    // grid
            threadsPerBlock, 1, 1,             // block
            0, nullptr,                        // shared memory and stream
            kernelArgs, nullptr)                // args
            );

        hipDeviceSynchronize();
    }

    // retrieve the updated positions and velocities
    hipMemcpy(h_bodies, d_bodies, size, hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_bodies);
}